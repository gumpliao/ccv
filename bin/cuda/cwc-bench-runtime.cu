#undef USE_DISPATCH // nvcc doesn't support libdispatch
extern "C" {
#include "ccv.h"
}
#include <ctype.h>
#define CASE_TESTS // so that we don't include public available methods
#include "../lib/cuda/cwc_convnet.cu"
#include "../lib/ccv_convnet.c"

extern "C" void cwc_bench_runtime(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_convnet_train_param_t params)
{
	int batch = params.mini_batch;
	int i;
	_cwc_convnet_alloc_reserved_both(convnet, batch, params.layer_params);
	cwc_convnet_context_t* context = GPU(convnet)->contexts;
	for (i = 0; i < convnet->rows * convnet->cols * convnet->channels; i++)
		convnet->mean_activity->data.f32[i] = 128;
	_cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, ccv_size(225, 225), convnet->rows, convnet->cols, convnet->channels, 1000, 0, batch, 0, batch, context->host.input, context->host.c);
	hipMemcpy(context->device.input, context->host.input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * batch, hipMemcpyHostToDevice);

	hipEvent_t overallStart;
	hipEvent_t overallStop;
	hipEventCreate(&overallStart);
	hipEventCreate(&overallStop);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;
	VARY(GPU(convnet)->layers + 0)->convolutional.forward.x = 4;
	VARY(GPU(convnet)->layers + 0)->convolutional.forward.y = 8;
	VARY(GPU(convnet)->layers + 0)->convolutional.forward.z = 32;
	VARY(GPU(convnet)->layers + 3)->convolutional.forward.x = 4;
	VARY(GPU(convnet)->layers + 3)->convolutional.forward.y = 8;
	VARY(GPU(convnet)->layers + 3)->convolutional.forward.z = 32;
	VARY(GPU(convnet)->layers + 6)->convolutional.forward.x = 4;
	VARY(GPU(convnet)->layers + 6)->convolutional.forward.y = 8;
	VARY(GPU(convnet)->layers + 6)->convolutional.forward.z = 32;
	VARY(GPU(convnet)->layers + 7)->convolutional.forward.x = 4;
	VARY(GPU(convnet)->layers + 7)->convolutional.forward.y = 8;
	VARY(GPU(convnet)->layers + 7)->convolutional.forward.z = 32;
	VARY(GPU(convnet)->layers + 8)->convolutional.forward.x = 4;
	VARY(GPU(convnet)->layers + 8)->convolutional.forward.y = 8;
	VARY(GPU(convnet)->layers + 8)->convolutional.forward.z = 32;
	hipEventRecord(overallStart, context->device.stream);
	for (i = 0; i < convnet->count; i++)
	{
		ccv_convnet_layer_t* layer = GPU(convnet)->layers + i;
		hipEventRecord(start, context->device.stream);
		_cwc_convnet_layer_forward_propagate(layer, i, layer->input.matrix.rows, layer->input.matrix.cols, batch, 0, i == 0 ? context->device.input : GPU(convnet)->forwards[i - 1], GPU(convnet)->forwards[i], GPU(convnet)->denoms[i], GPU(convnet)->unit, context);
		hipEventRecord(stop, context->device.stream);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		if (layer->type == CCV_CONVNET_CONVOLUTIONAL)
			printf("%d %d %d, elapsed time for layer %d fprop: %f milliseconds\n", VARY(layer)->convolutional.forward.x, VARY(layer)->convolutional.forward.y, VARY(layer)->convolutional.forward.z, i + 1, elapsed_time);
		else
			printf("elapsed time for layer %d fprop: %f milliseconds\n", i + 1, elapsed_time);
	}
	hipEventRecord(overallStop, context->device.stream);
	hipEventSynchronize(overallStop);
	hipEventElapsedTime(&elapsed_time, overallStart, overallStop);
	printf("forward pass %f milliseconds\n", elapsed_time);

	VARY(GPU(convnet)->layers + 0)->convolutional.backward.coefficient.x = 1;
	VARY(GPU(convnet)->layers + 0)->convolutional.backward.coefficient.y = 3;
	VARY(GPU(convnet)->layers + 0)->convolutional.backward.coefficient.z = 1;
	VARY(GPU(convnet)->layers + 3)->convolutional.backward.coefficient.x = 4;
	VARY(GPU(convnet)->layers + 3)->convolutional.backward.coefficient.y = 4;
	VARY(GPU(convnet)->layers + 3)->convolutional.backward.coefficient.z = 16;
	VARY(GPU(convnet)->layers + 3)->convolutional.backward.gradient.x = 4;
	VARY(GPU(convnet)->layers + 3)->convolutional.backward.gradient.y = 6;
	VARY(GPU(convnet)->layers + 3)->convolutional.backward.gradient.z = 24;
	VARY(GPU(convnet)->layers + 6)->convolutional.backward.coefficient.x = 8;
	VARY(GPU(convnet)->layers + 6)->convolutional.backward.coefficient.y = 3;
	VARY(GPU(convnet)->layers + 6)->convolutional.backward.coefficient.z = 32;
	VARY(GPU(convnet)->layers + 6)->convolutional.backward.gradient.x = 4;
	VARY(GPU(convnet)->layers + 6)->convolutional.backward.gradient.y = 8;
	VARY(GPU(convnet)->layers + 6)->convolutional.backward.gradient.z = 32;
	VARY(GPU(convnet)->layers + 7)->convolutional.backward.coefficient.x = 8;
	VARY(GPU(convnet)->layers + 7)->convolutional.backward.coefficient.y = 3;
	VARY(GPU(convnet)->layers + 7)->convolutional.backward.coefficient.z = 32;
	VARY(GPU(convnet)->layers + 7)->convolutional.backward.gradient.x = 4;
	VARY(GPU(convnet)->layers + 7)->convolutional.backward.gradient.y = 8;
	VARY(GPU(convnet)->layers + 7)->convolutional.backward.gradient.z = 32;
	VARY(GPU(convnet)->layers + 8)->convolutional.backward.coefficient.x = 8;
	VARY(GPU(convnet)->layers + 8)->convolutional.backward.coefficient.y = 4;
	VARY(GPU(convnet)->layers + 8)->convolutional.backward.coefficient.z = 32;
	VARY(GPU(convnet)->layers + 8)->convolutional.backward.gradient.x = 4;
	VARY(GPU(convnet)->layers + 8)->convolutional.backward.gradient.y = 8;
	VARY(GPU(convnet)->layers + 8)->convolutional.backward.gradient.z = 32;
	float* a = 0;
	hipMalloc(&a, sizeof(float) * 1000 * batch);
	hipMemcpy(a, GPU(convnet)->forwards[convnet->count - 1], sizeof(float) * 1000 * batch, hipMemcpyDeviceToDevice);
	hipEventRecord(overallStart, context->device.stream);
	for (i = convnet->count - 1; i >= 0; i--)
	{
		ccv_convnet_layer_t* layer = GPU(convnet)->layers + i;
		ccv_convnet_layer_t* configuration = GPU(convnet)->configurations + i;
		hipEventRecord(start, context->device.stream);
		switch (layer->type)
		{
			case CCV_CONVNET_CONVOLUTIONAL:
				if (context->device.dor[i])
				{
					int out_rows, out_cols, out_partition;
					_ccv_convnet_layer_derive_output(layer, layer->input.matrix.rows, layer->input.matrix.cols, &out_rows, &out_cols, &out_partition);
					_cwc_kern_mute_neuron
					<<<out_rows * out_cols * layer->net.convolutional.count, batch, 0, context->device.stream>>>
					(i == convnet->count - 1 ? a : GPU(convnet)->backwards[i + 1], context->device.dor[i]);
				}
				_cwc_convnet_convolutional_backward_propagate(layer, batch, i == convnet->count - 1 ? a : GPU(convnet)->backwards[i + 1], GPU(convnet)->forwards[i], i > 0 ? GPU(convnet)->forwards[i - 1] : context->device.input, GPU(convnet)->backwards[i], configuration, GPU(convnet)->scratch, GPU(convnet)->unit, context->device.stream, context->device.cublas);
				assert(hipGetLastError() == hipSuccess);
				break;
			case CCV_CONVNET_FULL_CONNECT:
				if (context->device.dor[i])
					_cwc_kern_mute_neuron
					<<<layer->net.full_connect.count, batch, 0, context->device.stream>>>
					(i == convnet->count - 1 ? a : GPU(convnet)->backwards[i + 1], context->device.dor[i]);
				_cwc_convnet_full_connect_backward_propagate(layer, batch,  i == convnet->count - 1 ? a : GPU(convnet)->backwards[i + 1], GPU(convnet)->forwards[i], i > 0 ? GPU(convnet)->forwards[i - 1] : context->device.input, GPU(convnet)->backwards[i], GPU(convnet)->unit, configuration, context->device.stream, context->device.cublas);
				assert(hipGetLastError() == hipSuccess);
				break;
			case CCV_CONVNET_LOCAL_RESPONSE_NORM:
				_cwc_convnet_rnorm_backward_propagate(layer, batch, i == convnet->count - 1 ? a : GPU(convnet)->backwards[i + 1], GPU(convnet)->forwards[i], i > 0 ? GPU(convnet)->forwards[i - 1] : context->device.input, GPU(convnet)->denoms[i], GPU(convnet)->backwards[i], context->device.stream);
				assert(hipGetLastError() == hipSuccess);
				break;
			case CCV_CONVNET_MAX_POOL:
				_cwc_convnet_max_pool_backward_propagate(layer, batch, i == convnet->count - 1 ? a : GPU(convnet)->backwards[i + 1], GPU(convnet)->forwards[i], i > 0 ? GPU(convnet)->forwards[i - 1] : context->device.input, GPU(convnet)->backwards[i], context->device.stream);
				assert(hipGetLastError() == hipSuccess);
				break;
			case CCV_CONVNET_AVERAGE_POOL:
				_cwc_convnet_average_pool_backward_propagate(layer, batch, i == convnet->count - 1 ? a : GPU(convnet)->backwards[i + 1], GPU(convnet)->backwards[i], context->device.stream);
				assert(hipGetLastError() == hipSuccess);
				break;
		}
		hipEventRecord(stop, context->device.stream);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		if (layer->type == CCV_CONVNET_CONVOLUTIONAL)
			printf("%d %d %d, %d %d %d, elapsed time for layer %d bprop: %f milliseconds\n", VARY(layer)->convolutional.backward.coefficient.x, VARY(layer)->convolutional.backward.coefficient.y, VARY(layer)->convolutional.backward.coefficient.z, VARY(layer)->convolutional.backward.gradient.x, VARY(layer)->convolutional.backward.gradient.y, VARY(layer)->convolutional.backward.gradient.z, i + 1, elapsed_time);
		else
			printf("elapsed time for layer %d bprop: %f milliseconds\n", i + 1, elapsed_time);
	}
	hipEventRecord(overallStop, context->device.stream);
	hipEventSynchronize(overallStop);
	hipEventElapsedTime(&elapsed_time, overallStart, overallStop);
	printf("backward pass %f milliseconds\n", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(overallStart);
	hipEventDestroy(overallStop);
	hipFree(a);
}
